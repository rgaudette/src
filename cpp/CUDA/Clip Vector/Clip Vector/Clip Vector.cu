#include "hip/hip_runtime.h"
﻿#define _WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <Windows.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include "Clip Vector.cuh"
#include <cstdio>

//template <typename float, typename int>

__global__ void clip_vector_kernel(float * vector, int n_elements, float low, float high)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < n_elements)
  {
    if (vector[idx] < low)
    {
      vector[idx] = low;
    }
    if (vector[idx] > high)
    {
      vector[idx] = high;
    }
  }
}


//template <typename float, typename int>
void clip_vector(float * vector, int n_elements, float low, float high)
{
  hipError_t cuda_status;

  // Uncomment to use the nvidia profiler, also the profiler stop call down below
  hipProfilerStart();

  // Allocate memory on the device
  float * d_vector;
  cuda_status = hipMalloc((void **) & d_vector, n_elements * sizeof(float));
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMalloc returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  cuda_status = hipMemcpy(d_vector,
                           vector,
                           n_elements * sizeof(float),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  // Launch a kernel on the GPU with one thread for each element.
  int n_threads_per_block = 256;
  int n_blocks = n_elements / n_threads_per_block;
  if (n_blocks * n_threads_per_block < n_elements)
  {
    n_blocks++;
  }
  LARGE_INTEGER start_time;
  QueryPerformanceCounter(&start_time);

  clip_vector_kernel<<<n_blocks, n_threads_per_block >>>(d_vector, n_elements, low, high);

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching clip_vector_kernel!\n", cuda_status);
  }

  LARGE_INTEGER stop_time;
  QueryPerformanceCounter(&stop_time);
  LARGE_INTEGER frequency;
  QueryPerformanceFrequency(&frequency);
  auto period = stop_time.QuadPart - start_time.QuadPart;
  double period_secs = static_cast<double>(period) / frequency.QuadPart;
  printf("computation time (secs): %e\n", period_secs);

  cuda_status = hipMemcpy(vector,
                           d_vector,
                           n_elements * sizeof(float),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  // Uncomment to use the nvidia profiler
  hipProfilerStop();
}


__global__ void clip_vector_nc_kernel(float * vector, int n_elements, float low, float high)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (vector[idx] < low)
    {
      vector[idx] = low;
    }
    if (vector[idx] > high)
    {
      vector[idx] = high;
    }
}


//template <typename float, typename int>
void clip_vector_nc(float * vector, int n_elements, float low, float high)
{
  hipError_t cuda_status;

  // Uncomment to use the nvidia profiler, also the profiler stop call down below
  hipProfilerStart();

  // Allocate memory on the device
  float * d_vector;
  cuda_status = hipMalloc((void **)& d_vector, n_elements * sizeof(float));
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMalloc returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  cuda_status = hipMemcpy(d_vector,
                           vector,
                           n_elements * sizeof(float),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  // Launch a kernel on the GPU with one thread for each element.
  int n_threads_per_block = 256;
  int n_blocks = n_elements / n_threads_per_block;
  if (n_blocks * n_threads_per_block < n_elements)
  {
    n_blocks++;
  }
  LARGE_INTEGER start_time;
  QueryPerformanceCounter(&start_time);

  clip_vector_nc_kernel << <n_blocks, n_threads_per_block >> >(d_vector, n_elements, low, high);

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching clip_vector_kernel!\n", cuda_status);
  }

  LARGE_INTEGER stop_time;
  QueryPerformanceCounter(&stop_time);
  LARGE_INTEGER frequency;
  QueryPerformanceFrequency(&frequency);
  auto period = stop_time.QuadPart - start_time.QuadPart;
  double period_secs = static_cast<double>(period) / frequency.QuadPart;
  printf("computation time (secs): %e\n", period_secs);

  cuda_status = hipMemcpy(vector,
                           d_vector,
                           n_elements * sizeof(float),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  // Uncomment to use the nvidia profiler
  hipProfilerStop();
}

//template <typename float, typename int>
__global__ void clip_vector_kernel2(float * vector, int n_elements, float low, float high)
{
  int idx_start = blockIdx.x * blockDim.x;
  int idx_stop = idx_start + blockDim.x;
  for (int idx = idx_start; idx < idx_stop; ++idx)
  {
    if (vector[idx] < low)
    {
      vector[idx] = low;
    }
    if (vector[idx] > high)
    {
      vector[idx] = high;
    }
  }
}


//template <typename float, typename int>
void clip_vector2(float * vector, int n_elements, float low, float high)
{
  hipError_t cuda_status;

  // Uncomment to use the nvidia profiler, also the profiler stop call down below
  hipProfilerStart();

  // Allocate memory on the device
  float * d_vector;
  cuda_status = hipMalloc((void **)& d_vector, n_elements * sizeof(float));
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMalloc returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  cuda_status = hipMemcpy(d_vector,
                           vector,
                           n_elements * sizeof(float),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }

  // Launch a kernel on the GPU with one thread for each element.
  int n_threads_per_block = 256;
  int n_blocks = n_elements / n_threads_per_block;
  if (n_blocks * n_threads_per_block < n_elements)
  {
    n_blocks++;
  }
  LARGE_INTEGER start_time;
  QueryPerformanceCounter(&start_time);

  clip_vector_kernel2<<<n_blocks, n_threads_per_block >>>(d_vector, n_elements, low, high);

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching clip_vector_kernel!\n", cuda_status);
  }

  LARGE_INTEGER stop_time;
  QueryPerformanceCounter(&stop_time);
  LARGE_INTEGER frequency;
  QueryPerformanceFrequency(&frequency);
  auto period = stop_time.QuadPart - start_time.QuadPart;
  double period_secs = static_cast<double>(period) / frequency.QuadPart;
  printf("computation time (secs): %e\n", period_secs);

  cuda_status = hipMemcpy(vector,
                           d_vector,
                           n_elements * sizeof(float),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy returned error code %d before launching clip_vector_kernel!\n", cuda_status);
  }
  // Uncomment to use the nvidia profiler
  hipProfilerStop();
}

//template void clip_vector<float, int>(float * vector, int n_elements, float low, float high);
//template void clip_vector<float, __int64>(float * vector, __int64 n_elements, float low, float high);
//template void clip_vector2<float, int>(float * vector, int n_elements, float low, float high);
//template void clip_vector2<float, __int64>(float * vector, __int64 n_elements, float low, float high);
