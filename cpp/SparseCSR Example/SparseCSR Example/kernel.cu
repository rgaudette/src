#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <memory>
#include <stdio.h>
#include <vector>
#include "mkl_spblas.h"

using namespace std;

hipError_t addWithCuda(int * c, const int * a, const int * b, unsigned int size);

__global__ void addKernel(int * c, const int * a, const int * b)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}


template <typename T>
struct CompressedCoordinateSparseMatrix
{
  int m;
  int n;
  unique_ptr<vector<T>> data;
  unique_ptr<vector<int>> row_index;
  unique_ptr<vector<int>> column_index;

  CompressedCoordinateSparseMatrix(int m, int n, int n_non_zero_hint) :
    m(m),
    n(n),
    data(new vector<T>(n_non_zero_hint)),
    row_index(new vector<int>(n_non_zero_hint)),
    column_index(new vector<int>(n_non_zero_hint))
  {

  }

  ~CompressedCoordinateSparseMatrix()
  {
  }

  void add(int row, int column, T value)
  {
    data->push_back(value);
    row_index->push_back(row);
    column_index->push_back(column);
  }

  int get_nnz()
  {
    return static_cast<int>(data->size());
  }

};


void print_mkl_error(sparse_status_t status)
{
  switch (status)
  {
    case SPARSE_STATUS_NOT_INITIALIZED:
      printf("SPARSE_STATUS_NOT_INITIALIZED\n");
      break;
    case SPARSE_STATUS_ALLOC_FAILED:
      printf("SPARSE_STATUS_ALLOC_FAILED\n");
      break;
    case SPARSE_STATUS_INVALID_VALUE:
      printf("SPARSE_STATUS_INVALID_VALUE\n");
      break;
    case SPARSE_STATUS_EXECUTION_FAILED:
      printf("SPARSE_STATUS_EXECUTION_FAILED\n");
      break;
    case SPARSE_STATUS_INTERNAL_ERROR:
      printf("SPARSE_STATUS_INTERNAL_ERROR\n");
      break;
    case SPARSE_STATUS_NOT_SUPPORTED:
      printf("SPARSE_STATUS_NOT_SUPPORTED\n");
      break;
    default:
      printf("Unknown MKL error: %d\n", status);
  }

}


int main()
{
  // Create a simple sparse identity matrix
  auto coo_eye = CompressedCoordinateSparseMatrix<float>(5, 5, 5);
  coo_eye.add(0, 0, 1.0F);
  coo_eye.add(1, 1, 1.0F);
  coo_eye.add(2, 2, 1.0F);
  coo_eye.add(3, 3, 1.0F);
  coo_eye.add(4, 4, 1.0F);

  sparse_status_t status;
  sparse_matrix_t A;
  status = mkl_sparse_s_create_coo(& A,
                                   SPARSE_INDEX_BASE_ZERO,
                                   coo_eye.m,
                                   coo_eye.n,
                                   coo_eye.get_nnz(),
                                   & (*coo_eye.row_index.get())[0],
                                   & (*coo_eye.column_index.get())[0],
                                   & (*coo_eye.data.get())[0]);

  if (status == SPARSE_STATUS_SUCCESS)
  {
    printf("Created MKL sparse COO array\n");
  }
  else
  {
    print_mkl_error(status);
    exit(-1);
  }

  // Convert the array to CSR
  sparse_matrix_t A_csr;
  status = mkl_sparse_convert_csr(A, SPARSE_OPERATION_NON_TRANSPOSE, &A_csr);
  if (status == SPARSE_STATUS_SUCCESS)
  {
    printf("Created MKL sparse CSR array\n");
  }
  else
  {
    print_mkl_error(status);
    exit(-1);
  }

  sparse_index_base_t index_base;
  int n_rows;
  int n_cols;
  int * rows_start;
  int * rows_end;
  int * col_indx;
  float * values;
  status = mkl_sparse_s_export_csr(A_csr,
                                   & index_base,
                                   & n_rows,
                                   & n_cols,
                                   & rows_start,
                                   & rows_end,
                                   & col_indx,
                                   & values);

  if (status == SPARSE_STATUS_SUCCESS)
  {
    printf("Exported MKL sparse CSR array\n");
  }
  else
  {
    print_mkl_error(status);
    exit(-1);
  }
  if (index_base != SPARSE_INDEX_BASE_ZERO)
  {
    printf("Warning 1 based indexing returned, expected 0 based.\n");
  }

  printf("%d x %d\n", n_rows, n_cols);
  printf("rows_start: ");
  for (int i = 0; i < n_rows; i++)
  {
    printf("%d, ", rows_start[i]);
  }
  printf("\n");

  printf("rows_end: ");
  for (int i = 0; i < n_rows; i++)
  {
    printf("%d, ", rows_end[i]);
  }
  printf("\n");

  int nnz = rows_end[n_rows - 1] - rows_start[0];

  printf("col_idx: ");
  for (int i = 0; i < nnz; i++)
  {
    printf("%d, ", col_indx[i]);
  }
  printf("\n");

  printf("values: ");
  for (int i = 0; i < nnz; i++)
  {
    printf("%f, ", values[i]);
  }
  printf("\n");


  // Allocate the necessary elements of a Compressed Sparse
  const int arraySize = 5;
  const int a[arraySize] = { 1, 2, 3, 4, 5 };
  const int b[arraySize] = { 10, 20, 30, 40, 50 };
  int c[arraySize] = { 0 };

  // Add vectors in parallel.
  hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "addWithCuda failed!");
    return 1;
  }

  printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
         c[0], c[1], c[2], c[3], c[4]);

  // hipDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Nsight and Visual Profiler to show complete traces.
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
  }

  return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int * c, const int * a, const int * b, unsigned int size)
{
  int * dev_a = 0;
  int * dev_b = 0;
  int * dev_c = 0;
  hipError_t cudaStatus;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // Allocate GPU buffers for three vectors (two input, one output)    .
  cudaStatus = hipMalloc((void **)&dev_c, size * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void **)&dev_b, size * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  addKernel <<< 1, size>>>(dev_c, dev_a, dev_b);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

Error:
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);

  return cudaStatus;
}
